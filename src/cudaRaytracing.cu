#include "hip/hip_runtime.h"
#include "cudaRaytracing.cuh"
#include "core/visibilityTester.cuh"
#include "lights/pointLight.cuh"
#include "core/dstructs.cuh"

#include <iostream>

__global__
void render(float4* pixel, Instance** object_list, unsigned int N, Camera camera, point3 point_light, const int width, const int height)
{
    int maxDepth = 3;

    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((x >= width) || (y >= height)) return;
    int offset = x + y * blockDim.x * gridDim.x;

    //const float aspect_ratio = 16.0 / 9.0;
    const float aspect_ratio = static_cast<float>(width) / height;
    float t = 1.0f;
    float r = aspect_ratio * t;

    Ray ray = compute_ray(x, y, camera, r, t, width, height);

    SurfaceInteraction rec;
    bool hit = intersection(ray, object_list, N, rec);

    pixel[offset] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);

    if (hit) // if there is an intersection
    {
        // preallocate a buffer for placement new -> faster than dynamic allocation 
        MemoryManager memory;
        rec.compute_scattering_functions(memory);

        vec3 wi;
        bool in_shadow;
        VisibilityTester visibility(object_list, N);

        PointLight light(point_light);
        Spectrum I = light.sample_li(rec, visibility, wi, in_shadow);

        Spectrum color = shade(rec, wi, in_shadow);   // for more than 1 ls -> color += shade() and inside for loop

        // trace rays for specular reflection and refraction
        ray.direction = rec.wo;
        //for (int i = 0; i < maxDepth - 1; ++i)
        //{
        //    //color += specularReflect(ray, rec, object_list, N, light, memory);
        //    color += specularRefract(ray, rec, object_list, N, light, memory);
        //}
        color += specularBounces(rec, maxDepth, object_list, N, light, memory);

        pixel[offset] = make_float4(color.x, color.y, color.z, 1.0f);
    }    
}

void callRayTracingKernel(
    float4* d_pixel,
    Instance** object_list,
    unsigned int N,
    Camera camera,
    point3 point_light,
    const int width,
    const int height)
{
    dim3 thread_block(8, 8, 1);
    dim3 grid(width / thread_block.x, height / thread_block.y, 1);
    render<<< grid, thread_block >>>(d_pixel, object_list, N, camera, point_light, width, height);
    hipDeviceSynchronize();
}

__device__
Spectrum specularBounces(const SurfaceInteraction& isect, const int maxDepth, Instance** scene, const int& N, PointLight light, MemoryManager& memory)
{
    const int refraction_rays = (powf(2, maxDepth) - 2) / 2;

    Spectrum color(0.0f);
    Stack<Ray> refraction_stack(refraction_rays);
    Stack<int> depth_stack(maxDepth);
    Stack<Spectrum> fresnel_stack(refraction_rays);
    Stack<normal3> normal_stack(refraction_rays);

    int tree_depth = 1;
    bool continue_loop = true;

    Ray ray;

    vec3 wo = isect.wo;         // outgoing direction
    vec3 wr;                    // reflected/transmited direction
    vec3 wt;                    // transmited direction
    normal3 n = isect.n;        // normal at intersection point
    vec3 wi;                    // incident direction (either reflected/transmited) -- to be used in the loop

    // Specular reflection
    Spectrum f = isect.bsdf.sample_f(wo, wr, BxDFType::SPECULAR_REFLECTION);
    if (!f.isBlack() && fabs(dot(wr, n)) != 0.0f && tree_depth < maxDepth)
    {
        ray = Ray(isect.p + wr * 0.0001f, wr);

        // if there Specular refraction
        Spectrum f_t = isect.bsdf.sample_f(wo, wt, BxDFType::SPECULAR_REFRACTION);
        if (!f_t.isBlack() && fabs(dot(wt, n)) != 0.0f && tree_depth < maxDepth)
        {
            refraction_stack.push(Ray(isect.p + wt * 0.0001f, wt));
            depth_stack.push(tree_depth);
            fresnel_stack.push(f_t);
            normal_stack.push(n);
        }
    }

    while (continue_loop)
    {
        wi = ray.direction;
        if (!f.isBlack() && fabs(dot(wi, n)) != 0.0f && tree_depth < maxDepth)
        {
            SurfaceInteraction isect;
            bool hit = intersection(ray, scene, N, isect);

            if (hit) // if there is an intersection
            {
                tree_depth += 1;

                isect.n = dot(isect.n, ray.direction) < 0.0f ? -isect.n : isect.n;
                isect.compute_scattering_functions(memory);

                vec3 wi;  // light direction coming from the light source
                bool in_shadow;
                VisibilityTester visibility(scene, N);

                Spectrum I = light.sample_li(isect, visibility, wi, in_shadow);

                color += f * shade(isect, wi, in_shadow) * fabs(dot(ray.direction, n));

                // ------------------------------------------------------------------
                vec3 wo = isect.wo;         // outgoing direction
                vec3 wr;                    // reflected/transmited direction
                vec3 wt;                    // transmited direction
                n = isect.n;                // normal at intersection point

                // Specular reflection
                Spectrum f = isect.bsdf.sample_f(wo, wr, BxDFType::SPECULAR_REFLECTION);
                if (!f.isBlack() && fabs(dot(wr, n)) != 0.0f)
                {
                    ray = Ray(isect.p + wr * 0.0001f, wr);

                    // Specular refraction
                    Spectrum f_t = isect.bsdf.sample_f(wo, wt, BxDFType::SPECULAR_REFRACTION);
                    if (!f_t.isBlack() && fabs(dot(wt, n)) != 0.0f)
                    {
                        refraction_stack.push(Ray(isect.p + wt * 0.0001f, wt));
                        depth_stack.push(tree_depth);
                        fresnel_stack.push(f_t);
                        normal_stack.push(n);
                    }
                }
                else
                    continue_loop = false;
                // -------------------------------------------------------------------
            }
            else
                continue_loop = false;
        }
        else
        {
            color += Spectrum(0.0f);
            continue_loop = false;
        }

        if (!continue_loop && !refraction_stack.isEmpty() && 
            !depth_stack.isEmpty() && 
            !fresnel_stack.isEmpty() && 
            !normal_stack.isEmpty())
        {
            ray = refraction_stack.pop();
            tree_depth = depth_stack.pop();
            f = fresnel_stack.pop();
            n = normal_stack.pop();
            continue_loop = true;
        }
    }
    return color;
}

__device__
Spectrum specularReflect(Ray& ray, SurfaceInteraction& isect, Instance** scene, const int& N, PointLight light, MemoryManager& memory)
{
    vec3 wo = isect.wo;
    vec3 wi;  // specular reflected direction

    normal3 n = isect.n;
    BxDFType type = BxDFType::SPECULAR_REFLECTION;
    Spectrum f = isect.bsdf.sample_f(wo, wi, type);
    
    if (!f.isBlack() && fabs(dot(wi, n)) != 0.0f)
    {
        ray = Ray(isect.p + wi * 0.0001f, wi);

        SurfaceInteraction t_isect;
        bool hit = intersection(ray, scene, N, t_isect);

        // clear bsdf array before adding new BxDFs
        isect.bsdf.clear();
        isect = t_isect;

        if (hit) // if there is an intersection
        {
            isect.n = dot(isect.n, ray.direction) < 0.0f ? -isect.n : isect.n;
            isect.compute_scattering_functions(memory);

            vec3 wi;  // light direction coming from the light source
            bool in_shadow;
            VisibilityTester visibility(scene, N);

            Spectrum I = light.sample_li(isect, visibility, wi, in_shadow);

            Spectrum color = f * shade(isect, wi, in_shadow) * fabs(dot(ray.direction, n));
            return color;
        }
        else
            return Spectrum(0.0f);
    }
    else
        return Spectrum(0.0f);

}

__device__
Spectrum specularRefract(Ray& ray, SurfaceInteraction& isect, Instance** scene, const int& N, PointLight light, MemoryManager& memory)
{
    vec3 wo = isect.wo;
    vec3 wi;            // specular refracted direction

    normal3 n = isect.n;   
    BxDFType type = BxDFType::SPECULAR_REFRACTION;
    Spectrum f = isect.bsdf.sample_f(wo, wi, type);
    
    if (!f.isBlack() && fabs(dot(wi, n)) != 0.0f)
    {
        ray = Ray(isect.p + wi * 0.0001f, wi);
        
        SurfaceInteraction t_isect;
        bool hit = intersection(ray, scene, N, t_isect);

        // clear bsdf array before adding new BxDFs
        isect.bsdf.clear();
        isect = t_isect;

        if (hit) // if there is an intersection
        {
            // ensure the normal is always pointing outside of the object
            isect.n = dot(isect.n, ray.direction) < 0.0f ? -isect.n : isect.n;

            isect.compute_scattering_functions(memory);

            vec3 wi;  // light direction coming from the light source
            bool in_shadow;
            VisibilityTester visibility(scene, N);

            Spectrum I = light.sample_li(isect, visibility, wi, in_shadow);

            Spectrum color = f * shade(isect, wi, in_shadow) * fabs(dot(ray.direction, n));
            return color;

        }
        else
            return Spectrum(0.0f);

    }
    else
        return Spectrum(0.0f);
}