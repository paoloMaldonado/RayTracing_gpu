#include "hip/hip_runtime.h"
#include "reflection.cuh"

__device__
BSDF::~BSDF()
{
	for (int i = 0; i < num_bxdfs; ++i)
	{
		if (bxdfs[i])
		{
			bxdfs[i]->~BxDF();
		}
			
	}
	num_bxdfs = 0;
}

__device__
Spectrum BSDF::f(const vec3& wi, const vec3& wo) const
{
	Spectrum f(0.0f);
	for (int i = 0; i < num_bxdfs; ++i)
	{
		f += bxdfs[i]->f(wi, wo);
	}
	return f;
}

__device__ 
Spectrum BSDF::sample_f(const vec3& wo, vec3& wi, const BxDFType& type) const
{
	Spectrum f(0.0f);
	BxDF* bxdf = nullptr;
	for (int i = 0; i < num_bxdfs; ++i)
	{
		if (bxdfs[i]->type == type)
		{
			bxdf = bxdfs[i];
			break;
		}	
	}
	if (bxdf != nullptr) { f = bxdf->sample_f(wo, wi); }
	return f;
}

__device__ 
bool refract(const normal3& n, const vec3& wi, const float& etaRatio, vec3& wo)
{
	float cosThetaI = dot(wi, n);
	// compute cosThetaT using Snell's Law
	float sin2ThetaI = fmax(0.0f, 1.0f - cosThetaI * cosThetaI);
	float sin2ThetaT = etaRatio * etaRatio * sin2ThetaI;
	// check for total internal reflection (TIR)
	if (sin2ThetaT >= 1.0f)
		return false; // no refraction
	
	float cosThetaT = sqrtf(1.0f - sin2ThetaT);

	wo = etaRatio * -wi + (etaRatio * cosThetaI - cosThetaT) * n;
	return true;
}

__device__
float reflectanceFresnel(float cosThetaI, float etaI, float etaT)
{
	cosThetaI = clamp(cosThetaI, -1.0f, 1.0f);
	// determine which index is incident and which is refracted
	bool entering = cosThetaI > 0.0f;
	if (!entering)
	{
		swap(etaI, etaT);
		cosThetaI = fabs(cosThetaI);
	}
	// compute cosThetaT using Snell's Law
	float sinThetaI = sqrtf(fmax(0.0f, 1.0f - (cosThetaI * cosThetaI)));
	float sinThetaT = (etaI / etaT) * sinThetaI;
	// check for total internal reflection (TIR)
	if (sinThetaT >= 1.0f)
		return 1.0f; // no refraction

	float costThetaT = sqrtf(fmax(0.0f, 1 - (sinThetaT * sinThetaT)));

	float Rperp = (etaI * cosThetaI - etaT * costThetaT) /
				  (etaI * cosThetaI + etaT * costThetaT);
	float Rparl = (etaT * cosThetaI - etaI * costThetaT) /
				  (etaT * cosThetaI + etaI * costThetaT);

	return (Rperp * Rperp + Rparl * Rparl) / 2.0f;
}

__device__ float FresnelDielectric::evaluate(float cosThetaI) const
{
	return reflectanceFresnel(cosThetaI, etaI, etaT);
}

